#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <math.h>
#include <hip/hip_runtime.h>
extern "C"
{
	#include "randomInts.h"
}

#define MIN(x,y) ((x<y)?x:y)
#define MAX(x,y) ((x>y)?x:y)

typedef struct {
    int* counts;
} result;


__device__ int d_next_pow2(int n)
{
    int nBits = 0;
	while( (n>>nBits) > 0 )
		nBits++;
    return 1<<nBits;
}

__device__ inline void d_reduce_add_loop(int * B, int myId)
{
	int n = d_next_pow2(blockDim.x);
	for(int s=n/2; s > 0; s>>=1)
	{
		if( (threadIdx.x<s) && (threadIdx.x+s)<blockDim.x)
		{
			B[myId] += B[myId+s];
		}
		__syncthreads();
	}
}


// A and B should be same size
__global__ void range_count_kernel(int * count, int * A)
{
    int myId = threadIdx.x + blockIdx.x * blockDim.x;
	int myA = A[myId];

	for(int rangeBin=0; rangeBin<10; rangeBin++)
	{
		A[myId] = ((myA/100)==rangeBin);
		__syncthreads();
		d_reduce_add_loop(A, myId);		
		if(threadIdx.x==0)
		{
			count[blockIdx.x + gridDim.x*rangeBin] = A[myId];
		}
	}

}

__global__ void reduce_add_kernel(int * B, int * A, int rangeBin)
{
    int myId = threadIdx.x + blockIdx.x * blockDim.x;

	__syncthreads();
	d_reduce_add_loop(A, myId);
	if(threadIdx.x==0)
	{
		B[blockIdx.x + rangeBin*gridDim.x] = A[myId];
	}
}


result range_count_cuda(int *a, int n)
{

// allocate device memory
    int *d_A;
    hipMalloc((int**) &d_A, sizeof(int)*n);

// copy input array to device
    hipMemcpy(d_A, a, n*sizeof(int), hipMemcpyHostToDevice);

// call kernel
    int threadsPerBlock = MIN(n,1024);
    int nBlocks = (n-1)/threadsPerBlock + 1;
	nBlocks = MAX(1,nBlocks);

	int *d_all_counts;
    hipMalloc((int**) &d_all_counts, sizeof(int)*10*nBlocks);

// block level kernel call
	range_count_kernel<<<nBlocks,threadsPerBlock>>>(d_all_counts,d_A);
	hipDeviceSynchronize();

// reduce block results
	int *d_counts;
	while( nBlocks > 1 )
	{
		int new_nBlocks = (nBlocks-1)/1024+1;
		threadsPerBlock = MIN(nBlocks,1024);

		hipMalloc((int**) &d_counts,10*new_nBlocks*sizeof(int));

		for(int rangeBin=0; rangeBin<10; rangeBin++)
			reduce_add_kernel<<<new_nBlocks,threadsPerBlock>>>(d_counts, d_all_counts+nBlocks*rangeBin, rangeBin);
		hipDeviceSynchronize();

		hipFree(d_all_counts);
		d_all_counts = d_counts;
		nBlocks = new_nBlocks;
	}

// copy result back to host
    int* counts = (int*)malloc(10*sizeof(int));
    hipMemcpy(counts, d_counts, 10*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_counts);

	result res = {counts};
    return res;

}



result range_count_seq(int* a, int n)
{
	int* b = (int*)calloc(10,sizeof(int));
    for(int i=0; i<n; i++)
      b[a[i]/100]++;
	
	result res = {b};
	return res;
}


int main(int argc, char** argv)
{

	int exp = 25;
	int n = 1<<exp;
	int* h_A = (int*)malloc(n*(sizeof(int)));

// make test array
	writeRandomFile(n, "inp.txt");
   	readIntsFromFile("inp.txt",n,h_A);

// get CUDA result
	result cudaResult = range_count_cuda(h_A, n);

// get sequential result
	result seqResult = range_count_seq(h_A, n);

// print results
	printf("n   SEQ      CUDA\n-----------------\n");
	for(int i=0; i<10; i++)
		printf("%d %8d %8d\n",i, seqResult.counts[i], cudaResult.counts[i]);

// free array memory
	free(h_A);
	free(seqResult.counts);
	free(cudaResult.counts);

    return 0;
}
