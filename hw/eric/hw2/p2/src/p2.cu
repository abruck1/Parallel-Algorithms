#include "hip/hip_runtime.h"
#include "p2.h"
extern "C"
{
	#include "randomInts.h"
}

#define MAX_THREADS 1024

typedef struct {
    int* histogram;
	int* scan;
} result;


//***********************************************
// Host calling functions

result range_count_cuda(int *a, int n, const int shared)
{

// allocate device memory
    int *d_A, *d_temp;
    hipMalloc((int**) &d_A, n*sizeof(int));
    hipMalloc((int**) &d_temp, 10*n*sizeof(int));
	hipMemset(d_temp, 0, 10*n*sizeof(int));

// copy input array to device
    hipMemcpy(d_A, a, n*sizeof(int), hipMemcpyHostToDevice);

// call kernel
    int threadsPerBlock = MIN(n,MAX_THREADS);
    int nBlocks = (n-1)/threadsPerBlock + 1;
	nBlocks = MAX(1,nBlocks);

	int *d_all_counts;
    hipMalloc((int**) &d_all_counts, sizeof(int)*10*nBlocks);

// block level kernel call
	dim3 blocks(nBlocks,10);
	if(shared)
		range_count_kernel_shared<<<blocks,threadsPerBlock,threadsPerBlock*sizeof(int)>>>(d_all_counts,d_A, n);
	else
		range_count_kernel_global<<<blocks,threadsPerBlock>>>(d_all_counts,d_temp,d_A, n);
	hipDeviceSynchronize();
    hipFree(d_A);
	hipFree(d_temp);

// reduce block results
	int *d_counts;
	do
	{
		int new_nBlocks = (nBlocks-1)/MAX_THREADS+1;
		threadsPerBlock = MIN(nBlocks,MAX_THREADS);

		hipMalloc((int**) &d_counts,10*new_nBlocks*sizeof(int));

		blocks = dim3(new_nBlocks,10);
		if(shared)
			reduce_add_kernel_shared<<<blocks,threadsPerBlock,threadsPerBlock*sizeof(int)>>>(d_counts, d_all_counts, nBlocks);
		else
			reduce_add_kernel_global<<<blocks,threadsPerBlock>>>(d_counts, d_all_counts, nBlocks);
		hipDeviceSynchronize();

		hipFree(d_all_counts);
		d_all_counts = d_counts;
		nBlocks = new_nBlocks;
	} while( nBlocks > 1 );


// copy result back to host
    int* histogram = (int*)malloc(10*sizeof(int));
    hipMemcpy(histogram, d_counts, 10*sizeof(int), hipMemcpyDeviceToHost);

// scan on the resulting histogram for CDF
// known data size n=10
	hs_scan_kernel<<<1,10,10*sizeof(int)>>>(d_counts,10);
	hipDeviceSynchronize();
    int* scan = (int*)malloc(10*sizeof(int));
    hipMemcpy(scan, d_counts, 10*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_counts);

	result res = {histogram, scan};
    return res;

}


result range_count_seq(int* a, int n)
{
	int* hist = (int*)calloc(10,sizeof(int));
	int* scan = (int*)calloc(10,sizeof(int));
	
    for(int i=0; i<n; i++)
      hist[a[i]/100]++;
	
	scan[0] = hist[0];
	for(int i=1; i<10; i++)
		scan[i] = scan[i-1] + hist[i];

	result res = {hist, scan};
	return res;
}


//***********************************************
// Main function

int main(int argc, char** argv)
{

// run through several times for run time stats
	
	const int NRUNS = 1;
    struct timeval t;
    gettimeofday(&t, NULL);
    srand(t.tv_usec);
    double exp = (26.0*( (double)rand()/(double)RAND_MAX));
    int n = (int)pow(2,exp); 
	int errCnt = 0;
	
	for(int irun=0; irun<NRUNS; irun++)
	{
		int* h_A = (int*)malloc(n*(sizeof(int)));

	// make test array
		writeRandomFile(n, "inp.txt");
	   	readIntsFromFile("inp.txt",n,h_A);

	// get global memory CUDA result
		result cudaGlobalResult = range_count_cuda(h_A, n, 0);

	// get shared memory CUDA result
		result cudaSharedResult = range_count_cuda(h_A, n, 1);

	// get sequential result
		result seqResult = range_count_seq(h_A, n);

	// print results
		printf("\nrun SEQ_H    CUDA_G_H CUDA_S_H  |  SEQ_CM  CUDA_G_CM  CUDA_S_CM\n");
		printf("---------------------------------------------------------\n");
		for(int i=0; i<10; i++)
		{
			printf("%d %8d ",i, seqResult.histogram[i]);
			printf("%8d ", cudaSharedResult.histogram[i]);
			printf("%8d", cudaSharedResult.histogram[i]);
			printf("   |  %8d ", seqResult.scan[i]);
			printf("%8d ", cudaSharedResult.scan[i]);
			printf("%8d", cudaSharedResult.scan[i]);
			if( seqResult.histogram[i]!=cudaGlobalResult.histogram[i] 
				|| seqResult.histogram[i]!=cudaSharedResult.histogram[i]
				|| seqResult.scan[i]!=cudaGlobalResult.scan[i]
				|| seqResult.scan[i]!=cudaSharedResult.scan[i])
			{
				printf ("  XXX");
				errCnt++;
			}
			printf("\n");
		}

	// free array memory
		free(h_A);
		free(seqResult.histogram);
		free(cudaGlobalResult.histogram);
		free(cudaSharedResult.histogram);
		free(cudaSharedResult.scan);
		free(cudaGlobalResult.scan);
	}

	printf("n = %d\nerrCnt = %d\n",n,errCnt);
    return 0;
}
