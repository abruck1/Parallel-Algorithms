#include "hip/hip_runtime.h"
#include "p3.h"
extern "C"
{
	#include "randomInts.h"
}

#define MAX_THREADS 4

enum repeatCheck { check, noCheck };

__device__ void d_hs_scan(int myId, int *A, int n, int initVal)
{
	if(myId==0)
		A[myId] += initVal;
	int myVal = A[myId];
	__syncthreads();

	for(int s=1; s<n; s*=2)
	{
		if(myId>=s)
			myVal += A[myId-s];
		__syncthreads();
		if(myId<n)
			A[myId] = myVal;
		__syncthreads();
	}
}


// perform a scan for radix sort
__global__ void radix_sort_kernel(int *A, int n, int nDigits)
{
	extern __shared__ int sdata[];
	int *left = sdata;
	int *right = sdata+n;
	int offset = blockIdx.x * blockDim.x;
	int tid = threadIdx.x;
	int myId = tid + offset;


	for(int iDigit=0; iDigit<nDigits; iDigit++)
	{
		int myVal = A[myId];
		int radix = 1<<iDigit;
		left[tid] = !(myVal&radix);
		right[tid] = !(left[tid]);
		__syncthreads();
	
	// scan
		d_hs_scan(tid, left, n, 0);
		d_hs_scan(tid, right, n, left[n-1]);

	// scatter
		int index = (myVal&radix)?(right[tid]-1):(left[tid]-1);
		A[index+offset] = myVal;
		__syncthreads();
	}
}



// this will merge two pieces of array A of size treeLevel*blockDim.x into d_out
__global__ void parallel_merge_kernel(int *d_out, int *A, int treeLevel)
{

    int mergeID = blockIdx.x/treeLevel;
    int n = treeLevel*blockDim.x;    

	int offset = (mergeID/(2*treeLevel))*2*n;
	int myInd = threadIdx.x +(blockIdx.x%treeLevel)*blockDim.x;
	//printf("threadID=%d, blockId=%d, mergeID=%d, offset=%d, myInd=%d\n",threadIdx.x, blockIdx.x, mergeID, offset, myInd);
	A = A+offset;
	int *B = A+n;

	// swap A and B if this is an odd block
	if(mergeID%2)
	{
		int *C = A;
		A = B;
		B = C;
	}
		
	int otherInd = d_binary_search(B, A[myInd], n);
	int mergedIndex = myInd + otherInd;
	int nRepeats=0;

	// sensitive to repeated elements if an odd block
	if(mergeID%2)
	{
	 	nRepeats = otherInd - d_binary_search(B,A[myInd]-1,n);		
		mergedIndex -= nRepeats;
	}

	d_out[mergedIndex+offset] = A[myInd];	
}




void cuda_radix_sort(int *A, int n, int nDigits)
{
	// pad array if less than a power of 2
	int np2 = next_pow2(n);
	int* h_A = (int*)calloc(np2,(sizeof(int)));
	memcpy(h_A,A,n*sizeof(int));

	// allocate and fill device memory
	int *d_A, *d_B;
	hipMalloc((int**)&d_A, np2*sizeof(int));
	hipMalloc((int**)&d_B, np2*sizeof(int));
	hipMemcpy(d_A, h_A, np2*sizeof(int), hipMemcpyHostToDevice);

	// device config
	int nBlocks = (np2-1)/MAX_THREADS+1;
	int threadsPerBlock = MIN(MAX_THREADS,np2);

	// launch radix kernel
	radix_sort_kernel<<<nBlocks,threadsPerBlock,2*threadsPerBlock*sizeof(int)>>>(d_A, threadsPerBlock, nDigits);
	hipDeviceSynchronize();

/*
	// print A
	printf("A: ");
		hipMemcpy(h_A, d_A, np2*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0; i<np2; i++)
	{
		if(i%MAX_THREADS==0 && i>0)
			printf(" | ");
		printf("%d, ",h_A[i]);
	}
	printf("\n");
*/
	// merge sorted blocks
	int cnt=1;
//	int nMerges = nBlocks;
	while((nBlocks/cnt)>1)
	{
//		printf("merging round %d\n",cnt, nBlocks);
		parallel_merge_kernel<<<nBlocks,threadsPerBlock>>>(d_B,d_A,cnt);
		hipDeviceSynchronize();
/*
	// print B
		printf("B: ");
		hipMemcpy(h_A, d_B, np2*sizeof(int), hipMemcpyDeviceToHost);
		for(int i=0; i<np2; i++)
		{
			if(i%(MAX_THREADS*cnt*2)==0 && i>0)
				printf(" | ");
			printf("%d, ",h_A[i]);
		}
		printf("\n");
*/
		cnt*=2;
//		nMerges /= 2;
    // swap A and B
        int *d_C = d_A;
        d_A = d_B;
		d_B = d_C;
	}

	// copy result and free memory
	hipMemcpy(A, d_A+(np2-n), n*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_A);	
	hipFree(d_B);	
	free(h_A);

}


void seq_radix_sort(int *A, int n, int nDigits)
{

}


int main()
{

	int MAX_EXP = 5;
    struct timeval t;
    gettimeofday(&t, NULL);
    srand(t.tv_usec);
    double exp = (MAX_EXP*( (double)rand()/(double)RAND_MAX));
    int n = (int)pow(2,exp); 
	
	printf("MAX_THREADS = %d\n",MAX_THREADS);
	printf("n = %d\n",n);

	// make test array
	int* A = (int*)malloc(n*(sizeof(int)));
	writeRandomFile(n, "inp.txt");
	readIntsFromFile("inp.txt",n,A);

	printf("\n");
	for(int i=0; i<n; i++)
		printf("%d, ",h_A[i]);
	printf("\n");

	cuda_radix_sort(A, n, 10);

	// print A
	printf("A: ");
	for(int i=0; i<n; i++)
		printf("%d, ",h_A[i]);
	printf("\n");


	printf("Array is %s sorted\n", (checkSorted(h_A,n)?"\b":"NOT"));

	free(A);

	
}

