#include "p3.h"
extern "C"
{
	#include "randomInts.h"
}

#define MAX_THREADS 1024



void cuda_radix_sort(int *A, int n, int nDigits)
{
	// pad array if less than a power of 2
	int np2 = next_pow2(n);
	int* h_A = (int*)calloc(np2,(sizeof(int)));
	memcpy(h_A,A,n*sizeof(int));

	// allocate and fill device memory
	int *d_A, *d_B;
	hipMalloc((int**)&d_A, np2*sizeof(int));
	hipMalloc((int**)&d_B, np2*sizeof(int));
	hipMemcpy(d_A, h_A, np2*sizeof(int), hipMemcpyHostToDevice);

	// device config
	int nBlocks = (np2-1)/MAX_THREADS+1;
	int threadsPerBlock = MIN(MAX_THREADS,np2);

	// launch radix kernel
	radix_sort_kernel<<<nBlocks,threadsPerBlock,2*threadsPerBlock*sizeof(int)>>>(d_A, threadsPerBlock, nDigits);
	hipDeviceSynchronize();

	// merge sorted blocks
	int cnt=1;
	while((nBlocks/cnt)>1)
	{
		parallel_merge_kernel<<<nBlocks,threadsPerBlock>>>(d_B,d_A,cnt);
		hipDeviceSynchronize();
		cnt*=2;
    // swap A and B
        int *d_C = d_A;
        d_A = d_B;
		d_B = d_C;
	}

	// copy result and free memory
	hipMemcpy(A, d_A+(np2-n), n*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_A);	
	hipFree(d_B);	
	free(h_A);

}


void seq_radix_sort(int *A, int n, int nDigits)
{
	int *B = (int*)malloc(n*sizeof(int));
	if(nDigits%2)
		nDigits++;
	
	for(int iDigit=0; iDigit<nDigits; iDigit++)
	{
		int radix = 1<<iDigit;
		int l = 0;
		int r = n-1;
		for(int i=0; i<n; i++)
		{
			if(!(A[i]&radix))
				B[l++] = A[i];
			if(A[n-i-1]&radix)
				B[r--] = A[n-i-1];
		}

		int *C = A;
		A = B;
		B = C;
	}

	free(B);

}

//***********************************************
// Main function

int main(int argc, char** argv)
{

	if(argc<2)
		printf("No input files specified\n\n");

	for(int i=1; i<argc; i++)
	{
		char* nextFile = argv[i];
		printf("\n***********************************\n");
		printf("Running p3 for file %s\n",nextFile);
		printf("***********************************\n\n");

	//  read array file
	   	randArray ra = readIntsFromFile(nextFile);
		int* A = ra.A;
		int n = ra.n;

	// call CUDA redix sort
		cuda_radix_sort(A, n, 10);

	// seq sort for comparison
	   	randArray rb = readIntsFromFile(nextFile);
		int* B = rb.A;
		seq_radix_sort(B,n,10);

	// print some results
		printf("CUDA result: ");
		for(int i=0; i<MIN(n,15); i++)
			printf("%d, ",A[i]);
		printf("\b\b ...\n");
		printf("SEQ  result: ");
		for(int i=0; i<MIN(n,15); i++)
			printf("%d, ",A[i]);
		printf("\b\b ...\n");

	// check results
		printf("\n\nArray is %s sorted\n", (checkSorted(A,n)?"\b":"NOT"));

		int errCnt=0;
		for(int i=0; i<n; i++)
		{
			if(A[i]!=B[i])
			{
				printf("SORT disagreement at index %d\n",i);
				errCnt++;
			}	
		}
		printf("CUDA result matches sequential result\n\n");
		printf("n = %d\n",n);

		free(A);
		free(B);
	}
	return 0;
}

